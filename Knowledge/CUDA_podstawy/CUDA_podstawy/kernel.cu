#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <chrono>

// CUDA runtime
#include <hip/hip_runtime.h>
#include ""

//
//Host -> CPU
//magistrala
//Device -> GPU
//Grid:
//Block1 Block2 Block3 ... BlockN
//
//Block1:
//Thread1 Thread2 Thread3  ... ThreadN

// Kernel function to add the elements of two arrays
//__global__ wskazuje �e ta funkcja jest wywo�ywana na 
//GPU(device), i mo�e by� wywo�ywana przez kod z CPU(host)
__global__ void add1(int n, float* x, float* y)
{
    //0 1 2 '3' 4 5
    // blockDim.x =3 
    //1
    //0
	
	//blockDim.x = ilo�� w�tk�w w ramach jednego bloku
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n) { y[idx] = x[idx] + y[idx]; }
}

__global__ void add2(int n, float* x, float* y)
{
    for (int i = 0; i < n; ++i)
        y[i] = x[i] + y[i];
}

//Old method
void addOldMethod(int n, float* x, float* y)
{
    for (int i = 0; i < n; ++i)
        y[i] = x[i] + y[i];
}

int main(void)
{
    const int N = 1000000;

    float* x, * y;

    std::cout << "New method1:" << std::endl;

    // Allocate Unified Memory � accessible from CPU or GPU
    //Alokuje pamiec w Unified Memory, i zwraca do niej 
    //wskznik, do tej pamieci maja dost�p GPU i CPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    //to jest takie wolne poniewa� tablice inicjalizuje CPU a 
    //korzysta z niej GPU i dlatego musi przenie�� dane do GPU i potem 
    //z powrotem do CPU

    int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    int gridSize;       // The actual grid size needed, based on input size 
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, add1, 0, N);
    // Round up according to array size 
    gridSize = (N + blockSize - 1) / blockSize;
    //977 1024

    std::cout << "How many blocks: " << gridSize << std::endl;
    std::cout << "How many threads: " << blockSize << std::endl;

    // Benchmark phase
    auto start = std::chrono::steady_clock::now();
    //Start code testing

    // Run add on the GPU
    //potr�jne ostre nawiasy oznaczaj� �e kod CPU(host) wywo�uje
    //kod GPU(device)
    //pierwszy parametr oznacza ile kopi funkcji "add" ma by�
    //wywo�ane, to znaczy ile blok�w ma stworzy�, a drugi parametr
    //ile w�tk�w ma dzia�a� w ramch tego bloku
    add1 << <gridSize, blockSize >> > (N, x, y);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    //End code testing
    auto end = std::chrono::steady_clock::now();
    auto usecs = std::chrono::duration_cast<std::chrono::duration<float,
        std::chrono::microseconds::period>>(end - start);
    std::cout << "Elapsed time: " << usecs.count() << " us." << std::endl;

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    std::cout << "New method2:" << std::endl;

    // Allocate Unified Memory � accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Benchmark phase
    start = std::chrono::steady_clock::now();
    //Start code testing


    add2 << <1, 1 >> > (N, x, y);

    hipDeviceSynchronize();

    //End code testing
    end = std::chrono::steady_clock::now();
    usecs = std::chrono::duration_cast<std::chrono::duration<float,
        std::chrono::microseconds::period>>(end - start);
    std::cout << "Elapsed time: " << usecs.count() << " us." << std::endl;

    // Check for errors (all values should be 3.0f)
    maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    //Old method
    std::cout << "Old method:" << std::endl;
    x = new float[N];
    y = new float[N];

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Benchmark phase
    start = std::chrono::steady_clock::now();
    //Start code testing

    // Run add on the CPU
    addOldMethod(N, x, y);

    //End code testing
    end = std::chrono::steady_clock::now();
    usecs = std::chrono::duration_cast<std::chrono::duration<float,
        std::chrono::microseconds::period>>(end - start);
    std::cout << "Elapsed time: " << usecs.count() << " us." << std::endl;

    // Check for errors (all values should be 3.0f)
    maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    //Free memory
    delete[] x;
    delete[] y;

    //int k;
    //std::cin >> k;
    return 0;
}

/*// Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    int howMany = 0;
    int OnElemen = 0;
    for (int i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
        if ((y[i] - 3.0f) != 0)
        {
            howMany += 1;
            OnElemen = i;
        }
    }
    std::cout << "Max error: " << maxError << std::endl;
    std::cout << "How many: " << maxError << std::endl;
    std::cout << "On element: " << OnElemen << std::endl;*/