
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

//Mno�enie macierzy
// 1 2 * 5 6 = (1*5+2*7) (1*6+2*8)
// 3 4   7 8   (3*5+4*7) (3*6+4*8)

__global__ void matrixMul(int* m, int* n, int* p, int size)
{	
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int p_sum = 0;

	for (int i = 0; i < size; ++i)
	{
		p_sum += m[row * size + i] * n[i * size + column];
	}

	p[row * size + column] = p_sum;
}

void matrixMul_seq(int* m, int* n, int* p, int size)
{
	for (int i = 0; i < size; ++i)
	{
		for (int j = 0; j < size; ++j)
		{
			for (int k = 0; k < size; ++k)
			{
				 p[i * size + j] += m[i * size + k] * n[k * size + j];
			}
		}
	}
}

int main()
{
	//W C/C++ jest Row-major//////
	/*int tab[2][2] = {{1,2},{3,4}};
	int* tab_ptr = &tab[0][0];
	for (int i = 0; i < 4; ++i)
	{
		std::cout << *tab_ptr << " ";
		++tab_ptr;
	}
	std::cout << std::endl;
	for (int i = 0; i < 2; ++i)
	{
		for (int j = 0; j < 2; ++j)
		{
			std::cout << tab[i][j] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;*/
	//////////////////////////////
	
	int n = 1 << 10; // == 1024 or 2^10
	printf("Square matrix of size %d\n", n);

	//n*m = p
	//Host Matrix  m, n, p
	int* h_m;
	int* h_n;
	int* h_p;
	int* h_p_seq;

	//Device Matrix  m, n, p
	int* d_m;
	int* d_n;
	int* d_p;

	size_t bytes = n * n * sizeof(int);

	//Allocate memory on host side
	h_m = (int*)malloc(bytes);
	h_n = (int*)malloc(bytes);
	h_p = (int*)malloc(bytes);
	h_p_seq = (int*)malloc(bytes);
	
	//Initialize matrix m, n, p
	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			h_m[i * n + j] = rand() % 1024;
			h_n[i * n + j] = rand() % 1024;
			h_p_seq[i * n + j] = 0;
		}
	}

	//Allocate memoru on device side
	hipMalloc(&d_m, bytes);
	hipMalloc(&d_n, bytes);
	hipMalloc(&d_p, bytes);

	//Copy data to the device
	hipMemcpy(d_m, h_m, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_n, h_n, bytes, hipMemcpyHostToDevice);

	int threads_per_block = 16;
	dim3 block_size(threads_per_block, threads_per_block);
	dim3 grid_size(n / block_size.x, n / block_size.y);
	
	printf("Grid size X: %d, Grid size Y: %d\n", grid_size.x, grid_size.y);
	printf("block_size X: %d, block_size Y: %d\n", block_size.x, block_size.y);

	// Benchmark phase
	auto start = std::chrono::steady_clock::now();
	//Start code testing
	matrixMul<<<grid_size, block_size>>> (d_m, d_n, d_p, n);
	hipDeviceSynchronize();
	//End code testing
	auto end = std::chrono::steady_clock::now();
	auto usecs = std::chrono::duration_cast<std::chrono::duration<float,
		std::chrono::milliseconds::period>>(end - start);
	std::cout << "Elapsed time: " << usecs.count() << " ms." << std::endl;

	// Benchmark phase
	start = std::chrono::steady_clock::now();
	//Start code testing
	matrixMul_seq(h_m, h_n, h_p_seq, n);
	//End code testing
	end = std::chrono::steady_clock::now();
	usecs = std::chrono::duration_cast<std::chrono::duration<float,
		std::chrono::milliseconds::period>>(end - start);
	std::cout << "Elapsed time: " << usecs.count() << " ms." << std::endl;

	hipMemcpy(h_p, d_p, bytes, hipMemcpyDeviceToHost);

	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			if(h_p[n * i + j] != h_p_seq[n * i + j])
			{
				printf("ERROR!\n");
				printf("h_p: %d, h_p_seq: %d\n", h_p[n * i + j], h_p_seq[n * i + j]);
			}
		}
	}

	free(h_m);
	free(h_n);
	free(h_p);
	free(h_p_seq);

	hipFree(d_m);
	hipFree(d_n);
	hipFree(d_p);

	return 0;
}

